
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 10000
// Kernel definition
__global__ 
void VecAdd(int* A, int* B, int* C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
    //printf("%i ",C[i]);
}

int main()
{
    int A[N],B[N],C[N],*d_a,*d_b,*d_c;
    int i;
    for(i=0;i<N;i++){
	A[i]=1;
	B[i]=1;
    }
    hipMalloc((void**)&d_a,N*sizeof(int));
    hipMalloc((void**)&d_b,N*sizeof(int)); 
    hipMalloc((void**)&d_c,N*sizeof(int));

    hipMemcpy(d_a,A,N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_b,B,N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_c,C,N*sizeof(int),hipMemcpyHostToDevice);
 
    VecAdd<<<(N/1024)+1, N>>>(d_a, d_b, d_c);

    //cudaMemcpy(A,d_a,N*sizeof(int),cudaMemcpyHostToDevice);
    //cudaMemcpy(B,d_b,N*sizeof(int),cudaMemcpyHostToDevice);
    hipMemcpy(C,d_c,N*sizeof(int),hipMemcpyDeviceToHost);     

    for(i=0;i<N;i++){
	printf("%i ",C[i]);
    }
    printf("\n");
    hipDeviceSynchronize();    
	
    return 0;
}
